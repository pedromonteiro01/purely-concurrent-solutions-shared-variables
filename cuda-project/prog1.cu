#include "hip/hip_runtime.h"
/**
 *  \file prog1.cu (implementation file)
 *
 *  \brief Problem name: Producers / Consumers.
 *
 *  Synchronization based on monitors.
 *  Both threads and the monitor are implemented using the pthread library which enables the creation of a
 *  monitor of Lampson / Redell type.
 *
 *  Generator thread of the intervening entities.
 *
 *  \author António Rui Borges - March 2023
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include <hip/hip_runtime.h>

/**
 *   program configuration
 */

#ifndef DIM
# define DIM 1024
#endif

/* allusion to internal functions */

/* returns the number of seconds elapsed between the two specified times */
static double get_delta_time(void);

/* returns 1 if the specified array is sorted, and 0 otherwise */
int validateSort(int *arr, int N);

/* Function to merge two haves of array */
__device__ void merge(int arr[], int l, int m, int r);

/* Iterative mergesort */
__device__ void mergeSort(int arr[], int n);

/* kernel function */
__global__ void processor(int *data, int iter);


/**
 *  \brief Function merge.
 *
 *  This function merges two sorted subarrays into a single sorted subarray.
 *
 *  \param arr: pointer to the array containing the subarrays
 *  \param l: starting index of the first subarray
 *  \param m: ending index of the first subarray and starting index of the second subarray
 *  \param r: ending index of the second subarray
 *
 *  The function creates temporary arrays to store the subarrays and then merges them into the original 
 *  array in a sorted order.
 */
__device__ void merge(int arr[], int l, int m, int r)
{
    int i, j, k;
    int n1 = m - l + 1;
    int n2 =  r - m;

	int *L = (int*)malloc(n1 * sizeof(int));
	int *R = (int*)malloc(n2 * sizeof(int));
 
    // Copy data to temp arrays
    for (i = 0; i < n1; i++)
        L[i] = arr[l + i];
    for (j = 0; j < n2; j++)
        R[j] = arr[m + 1+ j];
 
    // Merge temp arrays into arr
    i = 0;
    j = 0;
    k = l;
    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            arr[k] = L[i];
            i++;
        } else {
            arr[k] = R[j];
            j++;
        }
        k++;
    }
 
    // Copy remaining elements of L[]
    while (i < n1) {
        arr[k] = L[i];
        i++;
        k++;
    }
 
    // Copy remaining elements of R[]
    while (j < n2) {
        arr[k] = R[j];
        j++;
        k++;
    }

	free(L);
	free(R);
}

/**
 *  \brief Function mergeSort.
 *
 *  This function sorts an array using the merge sort algorithm.
 *
 *  \param array: pointer to the array to be sorted
 *  \param size: size of the array
 *
 *  The function divides the array into smaller subarrays and recursively sorts them using merge sort. 
 *  It then merges the sorted subarrays to obtain the final sorted array.
 */
__device__ void mergeSort(int array[], int size) {
   int currentSize, leftStart;
	
	for (currentSize = 1; currentSize <= size - 1; currentSize = 2 * currentSize) {
		for (leftStart = 0; leftStart < size - 1; leftStart += 2 * currentSize) {
           int middle = min(leftStart + currentSize - 1, size - 1);
           int rightEnd = min(leftStart + 2 * currentSize - 1, size - 1);
           merge(array, leftStart, middle, rightEnd);
       	}
   	}
}

/**
 *  \brief Function processor.
 *
 *  This CUDA kernel function performs parallel processing on the input array using merge sort algorithm.
 *
 *  \param data: pointer to the input array
 *  \param iter: iteration number indicating the level of merge sort
 *
 *  The function divides the input array into subsequences and sorts them using merge sort.
 *  Each thread is responsible for sorting a specific subsequence.
 *  In each iteration, the function performs either an independent merge sort on a subsequence (when iter is 0) 
 *  or merges two previously sorted subsequences.
 */
__global__ void processor(int *data, int iter) {
	int N = DIM;
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = blockDim.x * gridDim.x * y + x;

	if(idx >= (N >> iter)) return;

	int start = N * (1 << iter) * idx;
	int end = start + (1 << iter) * N;
	int mid = (start+end)/2;
	int subseq_len = (1 << iter) * N;
	int *subseq_start = data + start;

	(iter == 0) ? mergeSort(subseq_start, subseq_len) : merge(data, start, mid-1, end-1);
}


/**
 *  \brief Main function.
 *
 *  This function is the entry point of the program.
 *
 *  \param argc: number of command-line arguments
 *  \param argv: array of command-line argument strings
 *
 *  The function reads an input file containing integers, performs parallel merge sort using CUDA, 
 *  and validates the sorted array.
 */
int main (int argc, char **argv)
{
	if (argc != 2) {
		printf("Usage: %s <filename>\n", argv[0]);
		return 1;
	}

	/* Open the file for reading */

	FILE *file = fopen(argv[1], "rb");
	if (file == NULL) {
		printf("Failed to open file: %s\n", argv[1]);
		return 1;
	}

	fseek(file, 0, SEEK_END);
	int size = ftell(file) / sizeof(int);
	fseek(file, 0, SEEK_SET);

	int *host_matrix = (int*) malloc(size * sizeof(int));
	if (host_matrix == NULL) {
		printf("Error: cannot allocate memory\n");
		return 1;
	}

	int count = fread(host_matrix, sizeof(int), size, file);

	if (count != size) {
		printf("Error: could not read all integers from file\n");
		return 1;
	}

	fclose(file);


	/* set up the device */

	int dev = 0;

	hipDeviceProp_t deviceProp;
	CHECK (hipGetDeviceProperties (&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK (hipSetDevice (dev));

	/* copy the host data to the device memory */
	int *device_matrix;
	CHECK(hipMalloc((void**)&device_matrix, DIM * DIM * sizeof(int)));
	CHECK(hipMemcpy(device_matrix, host_matrix, DIM * sizeof(int[DIM]), hipMemcpyHostToDevice));


	/* launch the kernel */

	int gridDimX,gridDimY,gridDimZ,blockDimX,blockDimY,blockDimZ;

	// Number of threads in each dimension of a block
	blockDimX = 1 << 0;                                             // optimize!
	blockDimY = 1 << 0;                                             // optimize!
	blockDimZ = 1 << 0;                                             // do not change!

	// Number of blocks in each dimension of the grid
	gridDimX = 1 << 10;												// optimize!
	gridDimY = 1 << 0;												// optimize!
	gridDimZ = 1 << 0;                                              // do not change!

	dim3 grid (gridDimX, gridDimY, gridDimZ);
	dim3 block (blockDimX, blockDimY, blockDimZ);

	if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != DIM) {
		printf ("Wrong configuration!\n");
		printf("blockDimX = %d, blockDimY = %d, blockDimZ = %d\n", blockDimX, blockDimY, blockDimZ);
		printf("gridDimX = %d, gridDimY = %d, gridDimZ = %d\n", gridDimX, gridDimY, gridDimZ);
		return 1;
	}

	// Perform merge sort
	(void) get_delta_time ();

	for (int iter = 0; iter < 10; iter++) {
		processor<<<grid, block>>>(device_matrix, iter);
		blockDimX = DIM / (1 << (iter + 1));  // Divides by 2 each iteration
		dim3 block (blockDimX, blockDimY, blockDimZ);

		CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
		CHECK (hipGetLastError ());                                 // check for kernel errors
	}

	// Process one more iteration to merge the two halves (without updating the grid and block dimensions)
	processor<<<grid, block>>>(device_matrix, 10);

	
	CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
	CHECK (hipGetLastError ());                                 // check for kernel errors

	printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
			gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

	/* copy kernel result back to host side */
	CHECK (hipMemcpy (host_matrix, device_matrix, DIM * sizeof(int[DIM]), hipMemcpyDeviceToHost));

	/* free device global memory */
	CHECK (hipFree (device_matrix));

	/* reset the device */
	CHECK (hipDeviceReset ());

	// validate if the array is sorted correctly
	validateSort(host_matrix, DIM*DIM);
	free(host_matrix);
	return 0;
}

/**
 *  \brief Get delta time.
 *
 *  This function measures the elapsed time between successive calls.
 *
 *  \return The time elapsed between successive calls in seconds.
 *
 *  The function uses the CLOCK_MONOTONIC clock to measure time.
 */
static double get_delta_time(void)
{
	static struct timespec t0,t1;

	t0 = t1;
	if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
	{
		perror("clock_gettime");
		exit(1);
	}
	return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}

/**
 *  \brief Validate Sort.
 *
 *  This function checks if an array is sorted in ascending order.
 *
 *  \param arr: pointer to the array to be validated
 *  \param N: size of the array
 *
 */
int validateSort(int *arr, int N) {
    int i;

    for (i = 0; i < N - 1; i++)
    {
        if (arr[i] > arr[i + 1])
        {
            printf("Error in position %d between element %d and %d\n", i, arr[i], arr[i + 1]);
            return 0;
        }
    }
	if (i == (N - 1))
		printf("Everything is OK!\n");

    return 1;
}