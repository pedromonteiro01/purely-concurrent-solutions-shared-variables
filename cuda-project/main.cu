#include "hip/hip_runtime.h"
/**
 *   José Trigo
 *   Pedro Monteiro
 *   May 2023
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include <hip/hip_runtime.h>

/**
 *   program configuration
 */

#ifndef DIM
# define DIM 1024
#endif

/* allusion to internal functions */

static double get_delta_time(void);

__device__ void merge(int arr[], int l, int m, int r);
__device__ void mergeSort(int arr[], int n);
__global__ void processor(int *data, int iter);


/** \brief check if the array of integers has been sorted correctly */
int validateSort(int *arr, int N);


/* Function to merge the two haves arr[l..m] and arr[m+1..r] of array arr[] */
__device__ void merge(int arr[], int l, int m, int r)
{
    int i, j, k;
    int n1 = m - l + 1;
    int n2 =  r - m;

    /* create temp arrays */
	int *L = (int*)malloc(n1 * sizeof(int));
	int *R = (int*)malloc(n2 * sizeof(int));
 
    /* Copy data to temp arrays L[] and R[] */
    for (i = 0; i < n1; i++)
        L[i] = arr[l + i];
    for (j = 0; j < n2; j++)
        R[j] = arr[m + 1+ j];
 
    /* Merge the temp arrays back into arr[l..r]*/
    i = 0;
    j = 0;
    k = l;
    while (i < n1 && j < n2)
    {
        if (L[i] <= R[j])
        {
            arr[k] = L[i];
            i++;
        }
        else
        {
            arr[k] = R[j];
            j++;
        }
        k++;
    }
 
    /* Copy the remaining elements of L[], if there are any */
    while (i < n1)
    {
        arr[k] = L[i];
        i++;
        k++;
    }
 
    /* Copy the remaining elements of R[], if there are any */
    while (j < n2)
    {
        arr[k] = R[j];
        j++;
        k++;
    }

	free(L);
	free(R);
}

__device__ void mergeSort(int arr[], int n)
{
   int curr_size;  // For current size of subarrays to be merged
                   // curr_size varies from 1 to n/2
   int left_start; // For picking starting index of left subarray
                   // to be merged
	
   for (curr_size=1; curr_size<=n-1; curr_size = 2*curr_size)
   {
       for (left_start=0; left_start<n-1; left_start += 2*curr_size)
       {
           // Find ending point of left subarray. mid+1 is starting
           // point of right
           int mid = min(left_start + curr_size - 1, n-1);
 
           int right_end = min(left_start + 2*curr_size - 1, n-1);
 
           // Merge Subarrays arr[left_start...mid] & arr[mid+1...right_end]
           merge(arr, left_start, mid, right_end);
       }
   }
}

__global__ void processor(int *data, int iter) {
	int N = DIM;
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = blockDim.x * gridDim.x * y + x;

	if(idx >= (N >> iter)) return;

	int start = N * (1 << iter) * idx;
	int end = start + (1 << iter) * N;
	int mid = (start+end)/2;
	int subseq_len = (1 << iter) * N;
	int *subseq_start = data + start;
	
	if (iter ==0) {
		mergeSort(subseq_start, subseq_len);
	} else {
		if (idx == 0)
			printf("iter: %d, start: %d, mid: %d, end: %d\n", iter, start, mid, end);
		merge(data, start, mid-1, end-1);
	}
}


/**
 *   main program
 */

int main (int argc, char **argv)
{
	printf("%s Starting...\n", argv[0]);

	if (argc != 2) {
		printf("Usage: %s <filename>\n", argv[0]);
		return 1;
	}

	// Open the file for reading
	FILE *file = fopen(argv[1], "rb");
	if (file == NULL) {
		printf("Failed to open file: %s\n", argv[1]);
		return 1;
	}

	fseek(file, 0, SEEK_END);
	int size = ftell(file) / sizeof(int);
	fseek(file, 0, SEEK_SET);

	int *host_matrix = (int*) malloc(size * sizeof(int));
	if (host_matrix == NULL) {
		printf("Error: cannot allocate memory\n");
		return 1;
	}

	int count = fread(host_matrix, sizeof(int), size, file);

	if (count != size) {
		printf("Error: could not read all integers from file\n");
		return 1;
	}

	fclose(file);


	/* set up the device */
	int dev = 0;

	hipDeviceProp_t deviceProp;
	CHECK (hipGetDeviceProperties (&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK (hipSetDevice (dev));

	/* copy the host data to the device memory */
	int *device_matrix;
	(void) get_delta_time ();
	CHECK(hipMalloc((void**)&device_matrix, DIM * DIM * sizeof(int)));
	CHECK(hipMemcpy(device_matrix, host_matrix, DIM * sizeof(int[DIM]), hipMemcpyHostToDevice));
	printf ("The transfer of %ld bytes from the host to the device took %.3e seconds\n",
			DIM * sizeof(int[DIM]), get_delta_time ());

  	/* run the computational kernel
	 as an example, DIM threads are launched where each thread deals with one subsequence */

	int gridDimX,gridDimY,gridDimZ,blockDimX,blockDimY,blockDimZ;

	// Number of threads in each dimension of a block
	blockDimX = 1 << 10;                                             // optimize!
	blockDimY = 1 << 0;                                             // optimize!
	blockDimZ = 1 << 0;                                             // do not change!

	// Number of blocks in each dimension of the grid
	gridDimX = DIM / blockDimX;
	gridDimY = 1 << 0;
	gridDimZ = 1 << 0;                                              // do not change!

	dim3 grid (gridDimX, gridDimY, gridDimZ);
	dim3 block (blockDimX, blockDimY, blockDimZ);

	if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != DIM)
	{ printf ("Wrong configuration!\n");
	  return 1;
	}
  	(void) get_delta_time ();

	// Perform merge sort
	for (int iter = 0; iter < 11; iter++) {
		processor<<<grid, block>>>(device_matrix, iter);

		if (iter < 10) {
			blockDimX = DIM / (1 << (iter + 1));  // Divides by 2 each iteration
			gridDimX = DIM / blockDimX;
			dim3 block (blockDimX, blockDimY, blockDimZ);
			dim3 grid (gridDimX, gridDimY, gridDimZ);
		}

		CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
		CHECK (hipGetLastError ());                                 // check for kernel errors
	}

	printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
			gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

	/* copy kernel result back to host side */
	CHECK (hipMemcpy (host_matrix, device_matrix, DIM * sizeof(int[DIM]), hipMemcpyDeviceToHost));
	printf ("The transfer of %ld bytes from the device to the host took %.3e seconds\n",
			(long) DIM * sizeof(int[DIM]), get_delta_time ());

	/* free device global memory */
	CHECK (hipFree (device_matrix));

	/* reset the device */
	CHECK (hipDeviceReset ());

	//print array
	//for (int i = 0; i < DIM*DIM; i++)
	//	printf("%d ", host_matrix[i]);

	// validate if the array is sorted correctly
	validateSort(host_matrix, DIM*DIM);

	return 0;
}

static double get_delta_time(void)
{
	static struct timespec t0,t1;

	t0 = t1;
	if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
	{
		perror("clock_gettime");
		exit(1);
	}
	return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}

int validateSort(int *arr, int N)
{
	int i;

	for (i = 0; i < N - 1; i++)
	{
		if (arr[i] > arr[i + 1])
		{
			printf("Error in position %d between element %d and %d\n", i, arr[i], arr[i + 1]);
			return 0;
		}
		if (i == (N - 1))
			printf("Everything is OK!\n");
	}

	return 1;
}