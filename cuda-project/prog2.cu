#include "hip/hip_runtime.h"
/**
 *  \file prog2.cu
 *
 *  \brief Problem name: Int Sort Column processing.
 *
 *  \authors Pedro Monteiro & José Trigo - June 2023
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include <hip/hip_runtime.h>

/**
 *   program configuration
 */

#ifndef DIM
# define DIM 1024
#endif

/* allusion to internal functions */

/* returns the number of seconds elapsed between the two specified times */
static double get_delta_time(void);

/* returns 1 if the specified array is sorted, and 0 otherwise */
int validateSort(int *arr);

/* Function to merge two haves of array */
__device__ void merge(int arr[], int l, int m, int r, int idx, int iter);

/* Iterative mergesort */
__device__ void mergeSort(int array[], int size, int idx, int iter);

/* kernel function */
__global__ void processor(int *data, int iter);

/**
 *  \brief Function merge.
 *
 *  This function merges two sorted subarrays into a single sorted subarray within the device memory.
 *
 *  \param arr: pointer to the device array containing the subarrays
 *  \param l: starting index of the first subarray
 *  \param m: ending index of the first subarray and starting index of the second subarray
 *  \param r: ending index of the second subarray
 *  \param idx: index of the thread within the GPU grid
 *  \param iter: iteration number indicating the level of merge sort
 *
 */
__device__ void merge(int arr[], int l, int m, int r, int idx, int iter)
{
    int i, j, k;
    int n1 = m - l + 1;
    int n2 =  r - m;

    int *L = (int*)malloc(n1 * sizeof(int));
    int *R = (int*)malloc(n2 * sizeof(int));
 
    // Copy data to temp arrays
    for (i = 0; i < n1; i++)
        L[i] = arr[(1 << iter) * idx + DIM * ((l + i) % DIM) + ((l + i) / DIM)];
        // i - [ (i >> log2(N)) << log2(N) ]

    for (j = 0; j < n2; j++)
        R[j] = arr[(1 << iter) * idx + DIM * ((m + 1+ j) % DIM) + ((m + 1+ j) / DIM)];
 
    // Merge temp arrays into arr
    i = 0;
    j = 0;
    k = l;
    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            arr[(1 << iter) * idx + DIM * ((k) % DIM) + ((k) / DIM)] = L[i];
            i++;
        } else {
            arr[(1 << iter) * idx + DIM * ((k) % DIM) + ((k) / DIM)] = R[j];
            j++;
        }
        k++;
    }

    // Copy remaining elements of L[]
    while (i < n1) {
        arr[(1 << iter) * idx + DIM * ((k) % DIM) + ((k) / DIM)] = L[i];
        i++;
        k++;
    }
 
    // Copy remaining elements of R[]
    while (j < n2) {
        arr[(1 << iter) * idx + DIM * ((k) % DIM) + ((k) / DIM)] = R[j];
        j++;
        k++;
    }

    free(L);
    free(R);
}

/**
 *  \brief Function mergeSort.
 *
 *  This function performs merge sort on a subarray within the device memory.
 *
 *  \param array: pointer to the device array to be sorted
 *  \param size: size of the subarray
 *  \param idx: index of the thread within the GPU grid
 *  \param iter: iteration number indicating the level of merge sort
 *
 */
__device__ void mergeSort(int array[], int size, int idx, int iter) {
   int currentSize, leftStart;
    
    for (currentSize = 1; currentSize <= size - 1; currentSize = 2 * currentSize) {
        for (leftStart = 0; leftStart < size - 1; leftStart += 2 * currentSize) {
           int middle = min(leftStart + currentSize - 1, size - 1);
           int rightEnd = min(leftStart + 2 * currentSize - 1, size - 1);
           merge(array, leftStart, middle, rightEnd, idx, iter);
           }
       }
}

/**
 *  \brief Function processor.
 *
 *  This CUDA kernel function performs parallel processing on the input array using merge sort algorithm.
 *
 *  \param data: pointer to the input array
 *  \param iter: iteration number indicating the level of merge sort
 *
 */
__global__ void processor(int *data, int iter) {
	int N = DIM;
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = blockDim.x * gridDim.x * y + x;

	if(idx >= (N >> iter)) return;

	int start = 0;
	int end = start + (1 << iter) * N;
	int mid = (start+end)/2;
	int subseq_len = (1 << iter) * N;
	
	(iter == 0) ? mergeSort(data, subseq_len, idx, iter) : merge(data, start, mid-1, end-1, idx, iter);
}


/**
 *  \brief Main function.
 *
 *  This function is the entry point of the program.
 *
 *  \param argc: number of command-line arguments
 *  \param argv: array of command-line argument strings
 *
 *  The function reads an input file containing integers, performs merge sort using CUDA, 
 *  and validates the sorted array.
 */
int main (int argc, char **argv)
{
	if (argc != 2) {
		printf("Usage: %s <filename>\n", argv[0]);
		return 1;
	}

	/* Open the file for reading */

	FILE *file = fopen(argv[1], "rb");
	if (file == NULL) {
		printf("Failed to open file: %s\n", argv[1]);
		return 1;
	}

	fseek(file, 0, SEEK_END);
	int size = ftell(file) / sizeof(int);
	fseek(file, 0, SEEK_SET);

	int *host_matrix = (int*) malloc(size * sizeof(int));
	if (host_matrix == NULL) {
		printf("Error: cannot allocate memory\n");
		return 1;
	}

	int count = fread(host_matrix, sizeof(int), size, file);

	if (count != size) {
		printf("Error: could not read all integers from file\n");
		return 1;
	}

	fclose(file);

	/* set up the device */
	int dev = 0;

	hipDeviceProp_t deviceProp;
	CHECK (hipGetDeviceProperties (&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK (hipSetDevice (dev));

	/* copy the host data to the device memory */
	int *device_matrix;
	CHECK(hipMalloc((void**)&device_matrix, DIM * DIM * sizeof(int)));
	CHECK(hipMemcpy(device_matrix, host_matrix, DIM * sizeof(int[DIM]), hipMemcpyHostToDevice));

	/* launch the kernel */

	int gridDimX,gridDimY,gridDimZ,blockDimX,blockDimY,blockDimZ;

	// Number of threads in each dimension of a block
	blockDimX = 1 << 0;                                             // optimize!
	blockDimY = 1 << 0;                                             // optimize!
	blockDimZ = 1 << 0;                                             // do not change!

	// Number of blocks in each dimension of the grid
	gridDimX = 1 << 10;												// optimize!
	gridDimY = 1 << 0;												// optimize!
	gridDimZ = 1 << 0;                                              // do not change!

	dim3 grid (gridDimX, gridDimY, gridDimZ);
	dim3 block (blockDimX, blockDimY, blockDimZ);

	if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != DIM) {
		printf ("Wrong configuration!\n");
		printf("blockDimX = %d, blockDimY = %d, blockDimZ = %d\n", blockDimX, blockDimY, blockDimZ);
		printf("gridDimX = %d, gridDimY = %d, gridDimZ = %d\n", gridDimX, gridDimY, gridDimZ);
		return 1;
	}

	// Perform merge sort
	(void) get_delta_time ();

	for (int iter = 0; iter < 10; iter++) {
		processor<<<grid, block>>>(device_matrix, iter);
		gridDimX = DIM / (1 << (iter + 1));  // Divides by 2 each iteration
		dim3 grid (gridDimX, gridDimY, gridDimZ);

		CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
		CHECK (hipGetLastError ());                                 // check for kernel errors
	}

	// Process one more iteration to merge the two halves (without updating the grid and block dimensions)
	processor<<<grid, block>>>(device_matrix, 10);


	CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
	CHECK (hipGetLastError ());                                 // check for kernel errors

	printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
			gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

	/* copy kernel result back to host side */
	CHECK (hipMemcpy (host_matrix, device_matrix, DIM * sizeof(int[DIM]), hipMemcpyDeviceToHost));

	/* free device global memory */
	CHECK (hipFree (device_matrix));

	/* reset the device */
	CHECK (hipDeviceReset ());

	// validate if the array is sorted correctly
	validateSort(host_matrix);
	free(host_matrix);
	return 0;
}

/**
 *  \brief Get delta time.
 *
 *  This function measures the elapsed time between successive calls.
 *
 *  \return The time elapsed between successive calls in seconds.
 *
 *  The function uses the CLOCK_MONOTONIC clock to measure time.
 */
static double get_delta_time(void)
{
	static struct timespec t0,t1;

	t0 = t1;
	if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
	{
		perror("clock_gettime");
		exit(1);
	}
	return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}

/**
 *  \brief Validate Sort.
 *
 *  This function checks if a square matrix is sorted column-wise in ascending order.
 *
 *  \param arr: pointer to the array representing the square matrix
 *
 *  The function iterates through each column of the matrix and compares each element with the element below it.
 *
 */
int validateSort(int *arr){
    int N = 1024;

    for(int i=0; i < N; i++ ){
      for(int j=0; j < N-1; j++ ){
        if( arr[j*N+i] > arr[(j+1)*N+i] ){
          printf("Error in position %d between element %d and %d\n", i+j*N, arr[i+j*N], arr[(j+1)*N+i]);
          return 1;
        }
      }

      if(i == N-1){
        printf ("Everything is OK!\n");
        return 0;
        }

      if( arr[(N-1)*N+i] > arr[i+1] ){
        printf("Error in position %d between element %d and %d\n",(N-1)*N+i, arr[(N-1)*N+i],arr[i+1]);
        return 1;
      }
    }

    return 0;
}